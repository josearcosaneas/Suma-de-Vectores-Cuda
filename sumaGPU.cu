/*
    @autor José Arcos Aneas
    Codigo de suma de dos vectores contenidos en archivos donde la 
    primera linea sea el numero de elemento a leer.
    Los archivos son pasados como argumento a la hora de ejecutar.
*/

#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <sys/time.h>
using namespace std;

// Funcion de "nucleo" que calcula la suma dando a cada hebra
// la funcion de calcular la suma de los elemento de una posicion
__global__ void VecAdd(double* A,double* B,double* C)
{
    // extern __shared__ float sdata[];
	int i=threadIdx.x;
	C[i]=A[i]+B[i];

}

int main(int argc,char* argv[])
{
  // Leemos el fichero datos que cargamos desde entrada
  /*
  Esto es una actualizacion para no tener que compilar 
  con cada archivo que se quiera leer, no la he probado en 
  paralelo pero es identica a la secuencial y no ha dado 
  problemas.  
  */
  // leo el fichero 
  const char *fichero1 = argv[1];
  const char *fichero2 = argv[2];
  ifstream archivo1(fichero1);
  ifstream archivo2(fichero2);
  // varibles para calcular el tiempo
  struct timeval stop, start; 
  // leemos el numero de filas
  int filas,residuo;
  archivo2 >> residuo;
  archivo1 >> filas;
  cout << filas << "\n";
  //Leemos el contenido del fichero 
  float aux1,aux2=0.0;
  int i = 0;
  float lista1 [filas];
  float lista2 [filas];
  // todas la filas   
  for(i=0 ; i < filas; i++){
      archivo1 >> aux1;
      archivo2 >> aux2;      
      lista1[i]=aux1;
      lista2[i]=aux2;
  }   
	size_t size=filas*sizeof(double); 
	double* h_A=(double *)malloc(size);
	double* h_B=(double *)malloc(size);
	double* h_C=(double *)malloc(size);
	for(i=0;i<filas;i++)
	{
		h_A[i]=lista1[i];
		h_B[i]=lista2[i];
	}
    // Reservamos memoria y movemos las entradas a la memoria
    // del device
	double *d_A;
	hipMalloc((void**)&d_A,size);
	hipMemcpy(d_A,h_A,size,hipMemcpyHostToDevice);
	double *d_B;
	hipMalloc((void**)&d_B,size);
	hipMemcpy(d_B,h_B,size,hipMemcpyHostToDevice);
	double *d_C;
	hipMalloc((void**)&d_C,size);
	dim3 dimblock(filas);// si el numero es muy grande podriamos tener problemas ya que podria sobrepasar el maximo.
	// Inicia el nucleo para calcular la suma
    gettimeofday(&start, NULL);
	VecAdd<<<1,dimblock>>>(d_A,d_B,d_C);
    gettimeofday(&stop, NULL);
    // Copiamos el resultado del host
	hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost);
    // mostramos el resultado de la suma
	for(i=0;i<filas;i++)
	{
		printf("%lf ",h_C[i]);
	}
    // Liberamos memoria
	hipFree(d_A); 
	hipFree(d_B); 
	hipFree(d_C); 
	
	free(h_A); 
	free(h_B); 
	free(h_C);
    cout << "\nCodigo ejecutado en (ms) " << stop.tv_usec - start.tv_usec;
	// mostramos el tiempo empleado en ms
	//printf("\nCodigo Ejecutado en  %d(ms)\n",int(stop.tv_usec - start.tv_usec));
    return 0;
}
