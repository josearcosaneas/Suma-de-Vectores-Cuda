/*
    @autor José Arcos Aneas
    Archivo que muestra las caracteristicas de nuestros Devices
*/
#include "hip/hip_runtime.h"
#include ""
#include<stdio.h>
#include<stdlib.h>
#include<time.h>
int main() {
  int numeroDevice;
  hipGetDeviceCount(&numeroDevice);
  for (int i = 0; i < numeroDevice; i++) {
    cudaDevicepropiedades propiedades;
    cudaGetDevicepropiedadeserties(&propiedades, i);
    printf("Dispositivo Numero: %d\n", i);
    printf("  Nombre del dispositivo: %s\n", propiedades.name);
    printf("  Memory Clock Rate (KHz): %d\n",
           propiedades.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",
           propiedades.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
           2.0*propiedades.memoryClockRate*(propiedades.memoryBusWidth/8)/1.0e6);
  }
}
